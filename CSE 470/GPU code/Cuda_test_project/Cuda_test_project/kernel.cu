#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <stdio.h>

hipError_t Bellmanford(int *out, std::vector<int> V, std::vector<int> I, std::vector<int> E, std::vector<int> W, int blockSize);
hipError_t test(std::vector<int> V, std::vector<int> I, std::vector<int> E, std::vector<int> W, int times);


__global__ void arrayInit(int* a, int size, int pos);
__global__ void relax(int size, int* c_V, int* c_I, int* c_E, int* c_W, int* d_V, int* d_P, int* d_I);
__global__ void copy(int size, int* d_V, int* d_I);
__global__ void pred(int size, int* c_I, int* c_E, int* c_W, int* d_V, int* d_P);


int main() {
    int times = 1;
    std::vector<int> V = { 0, 1, 2 };
    std::vector<int> I = { 0, 2, 3, 4 };
    std::vector<int> E = { 1, 2, 2, 1};
    std::vector<int> W = { 0, 0, 3, -2 };
    if (test(V, I, E, W, times) != hipSuccess) return 1;

    V = {0, 1, 2, 3, 4};
    I = { 0, 2, 5, 6, 8, 10 };
    E = { 1, 3, 2, 3, 4, 1, 2, 4, 0, 2 };
    W = { 6, 7, 5, 8, -4, -2, -3, 9, 2, 7 };
    if (test(V, I, E, W, times) != hipSuccess) return 1;

    V = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19};
    I = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
    E = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 0};
    W = { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  -10};
    if (test(V, I, E, W, times) != hipSuccess) return 1;

    return 0;
}

hipError_t test(std::vector<int> V, std::vector<int> I, std::vector<int> E, std::vector<int> W, int times) {
    hipEvent_t start, stop;
    int out[20] = { 0 };
    float total = 0.0;
    int blockSize = 16;
    for (int i = 0; i < times; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        hipError_t cudaStatus = Bellmanford(out, V, I, E, W, blockSize);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Bellmanford failed!");
            return cudaStatus;
        }

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return cudaStatus;
        }
        total += milliseconds;
    }
    printf("{%d", out[0]);
    for (int i = 1; i < V.size(); i++) {
        printf(",%d", out[i]);
    }
    printf("}\n");
    total = total / times;
    printf("%d runs took %f ms\n", times, total);
    return hipSuccess;
}

hipError_t Bellmanford(int* out, std::vector<int> V, std::vector<int> I, std::vector<int> E, std::vector<int> W, int blockSize) {
    int *c_V, *c_I, *c_E, *c_W, *d_V, *d_P, *d_I;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&c_V, V.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&c_I, I.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&c_E, E.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&c_W, W.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_V, V.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_P, V.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_I, V.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c_V, V.data(), V.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        goto Error;
    }

    cudaStatus = hipMemcpy(c_I, I.data(), I.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        goto Error;
    }

    cudaStatus = hipMemcpy(c_E, E.data(), E.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        goto Error;
    }

    cudaStatus = hipMemcpy(c_W, W.data(), W.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed");
        goto Error;
    }

    int num_blocks = (V.size() + blockSize - 1) / blockSize;
    arrayInit <<<num_blocks, blockSize>>>(d_V, V.size(), 0);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "arrayInit launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    arrayInit <<<num_blocks, blockSize>>>(d_P, V.size(), 0);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "arrayInit launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    arrayInit<<<num_blocks, blockSize>>>(d_I, V.size(), 0);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "arrayInit launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    for (int i = 0; i < I.size() - 2; i++) {
        relax<<<num_blocks, blockSize>>>(V.size(), c_V, c_I, c_E, c_W, d_V, d_P, d_I);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "relax launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        copy<<<num_blocks, blockSize>>>(V.size(), d_V, d_I);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "copy launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
    }

    pred<<<num_blocks, blockSize>>>(V.size(), c_I, c_E, c_W, d_V, d_P);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "pred launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(out, d_V, V.size() * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(c_V);
    hipFree(c_I);
    hipFree(c_E);
    hipFree(c_W);
    hipFree(d_V);
    hipFree(d_P);
    hipFree(d_I);

    return cudaStatus;

}
__global__ void arrayInit(int *a, int size, int pos) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        a[index] = index == pos ? 0 : INT_MAX;
    }
}

__global__ void relax(int size, int* c_V, int* c_I, int* c_E, int* c_W, int* d_V, int* d_P, int* d_I) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        if (d_V[index] != INT_MAX) {
            for (int i = c_I[index]; i < c_I[index + 1]; i++) {
                int dis = d_V[index] + c_W[i];
                atomicMin(&d_I[c_E[i]], dis);
            }
        }
    }
}

__global__ void copy(int size, int* d_V, int* d_I) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        d_V[index] = d_I[index];
    }
}

__global__ void pred(int size, int* c_I, int* c_E, int* c_W, int* d_V, int* d_P) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        for (int i = c_I[index]; i < c_I[index + 1]; i++) {
            int dis = d_V[index] + c_W[i];
            if (dis <= d_V[c_E[i]]) {
                if (dis == d_V[c_E[i]]) {
                    d_P[c_E[i]] = index; // don't care which parent just that there is one
                }
            }
        }
    }
}


